#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include "../../comm/hip/hip_runtime_api.h"

__global__ void reduce(float* d_in,float *d_out) {
    unsigned int i=blockIdx.x*(blockDim.x*2)+threadIdx.x;
    unsigned int t=threadIdx.x;
    paartialSum[t]=d_in[i] + d_in[i+blockDim.x];
    __syncthreads();
    for(unsigned int stride=blockDim.x/2;stride>1;stride/=2){
        if(t<stride)
            partialSum[t]+=partialSum[t+stride];
        __syncthreads();
    }
    if(tid==0)d_out[blockIdx.x]=sdata[tid];
}

template<typename T>
void rand_array(T * array,size_t len){
    for(int i=0;i<len;++i){
        array[i]=((T)rand())/RAND_MAX;
    }
}
bool check(float* mat,float* res,int dim)
{
    for(int i=0;i<dim;++i){
        float p=0.0;
        for(int j=0;j<dim;++j)
            p+=mat[j];
        if(fabs(p-res[i])>1e-10) return false;
    }
    return true;
}

int main(){
    const size_t dim=300;
    size_t size=dim*dim*sizeof(float);
    float * mat=(float*)malloc(size);
    float * res=(float*)malloc(size/dim);
    if(mat==nullptr || res==nullptr){
        return -1;
    }
    rand_array<float>(mat,dim*dim);
    float * c_mat;
    float * c_res;
    checkCudaErrors(hipMalloc((void**)&c_mat, size));
    checkCudaErrors(hipMalloc((void**)&c_res, size/dim));
    checkCudaErrors(hipMemcpy(c_mat, mat, size, hipMemcpyHostToDevice));
    for(int i=0;i<dim;++i){
        reduce<<<ceil(dim/512),512>>>(c_mat+dim*i,c_res+i);
    }
    checkCudaErrors(hipMemcpy(res, c_res, size/dim, hipMemcpyDeviceToHost));
    if(!check(mat,res))
        std::cout<<"fail\n";
    checkCudaErrors(hipFree(c_mat));
    checkCudaErrors(hipFree(c_res));
    free(res);
    free(mat);
}