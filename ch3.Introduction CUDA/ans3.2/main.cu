#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include "../../comm/hip/hip_runtime_api.h"
__global__ void MatMulVec(float* ans,float* mat,float * vec,int dim) {
    int index=blockIdx.x*blockDim.x+threadIdx.x;
    if( index > dim*dim ) return;
    if( blockIdx.x>dim ) return;
    ans[blockIdx.x]+=mat[index]*vec[blockIdx.x];
}
template<typename T>
void rand_array(T * array,size_t len){
    for(int i=0;i<len;++i){
        array[i]=((T)rand())/RAND_MAX;
    }
}
void check(float* mat,float* vec,float* ans,size_t dim){
    float mans[100]={0.0};
    for(int i=0;i<dim;++i){
        for(int j=0;j<dim;++j){
            mans[j]+=mat[i*dim+j]*vec[j];
        }
    }
    for(int i=0;i<dim;++i){
        if(fabs(mans[i]-ans[i])>1e-10){
            printf("err");
        }
    }
}

int main(){
    const size_t dim=100;
    size_t size=dim*dim*sizeof(float);
    size_t vec_size=dim*sizeof(float);
    float* mat=(float*)malloc(size);
    float* vec=(float*)malloc(vec_size);
    float* ans=(float*)malloc(vec_size);
    if(mat==nullptr || vec==nullptr || ans==nullptr)
        return -1;
    rand_array<float>(mat,dim*dim);
    rand_array<float>(vec,dim);
    float* c_mat,*c_vec,*c_ans;
    checkCudaErrors(hipMalloc((void**)&c_mat, size));
    checkCudaErrors(hipMalloc((void**)&c_vec, vec_size));
    checkCudaErrors(hipMalloc((void**)&c_ans, vec_size));
    checkCudaErrors(hipMemcpy(c_mat, mat, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(c_vec, vec, vec_size, hipMemcpyHostToDevice));
    MatMulVec<<<ceil(dim / 256),256>>>(c_ans,c_mat,c_vec,dim);
    checkCudaErrors(hipMemcpy(ans, c_ans, vec_size, hipMemcpyDeviceToHost));
    check(c_mat,c_vec,c_ans,dim);
    checkCudaErrors(hipFree(mat));
    checkCudaErrors(hipFree(vec));
    checkCudaErrors(hipFree(ans));
    return 0;
}