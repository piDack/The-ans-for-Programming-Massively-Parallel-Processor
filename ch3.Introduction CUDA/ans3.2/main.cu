#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>


__global__ void MatMulVec(float* ans,float* mat,float * vec,int dim) {
    int index=blockIdx.x*blockDim.x+threadIdx.x;
    if( index > dim*dim ) return;
    if( blockIdx.x>dim ) return;
    ans[blockIdx.x]+=mat[index]*vec[blockIdx.x];
}

int main(){
    size_t dim=100;
    size_t size=dim*dim*sizeof(float);
    size_t vec_size=dim*sizeof(float);
    float* mat=(float*)malloc(size);
    float* vec=(float*)malloc(vec_size);
    float* ans=(float*)malloc(vec_size);
    float* c_mat,c_vec,c_ans;
    hipMalloc((void**)&c_mat, size);
    hipMalloc((void**)&c_mat, vec_size);
    hipMalloc((void**)&c_mat, vec_size);
    MatMulVec<<<ceil(dim / 256),256>>>(ans,mat,ans,dim);
    hipFree(mat);
    hipFree(vec);
    hipFree(ans);
    return 0;
}