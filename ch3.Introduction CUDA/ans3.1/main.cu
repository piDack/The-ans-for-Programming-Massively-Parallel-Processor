#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
__global__ void a(float *a,float * b,float *c,size_t n) {
    int i=blockIdx.y*blockDim.x+blockIdx.x;
    if(i > n*n) return;
    a[i]=b[i]+c[i];
}


__global__ void b(float *a,float * b,float *c,size_t n) {
    int offset=blockIdx.y*blockDim.x+blockIdx.x;
    if(offset > n) return;
    for(int i=0;i<dim;++i){
        a[i+offset]=b[i+offset]+c[i+offset];
    }
}


__global__ void c(float *a,float * b,float *c,size_t n) {
    int offset=blockIdx.y*blockDim.x+blockIdx.x;
    if(offset > n) return;
    for(int i=0;i<dim;++i){
        a[i*dim+offset]=b[i*dim+offset]+c[i*dim+offset];
    }
}

void stub(){
    size_t dim=100;
    size_t siz=dim*dim*sizeof(float);
    float* B=(float*)malloc(siz);
    float* A=(float*)malloc(siz);
    float* C=(float*)malloc(siz);
    float* c_A,c_B,c_C;
    hipMalloc((void**)&c_A, siz);
    hipMalloc((void**)&c_B, siz);
    hipMalloc((void**)&c_C, siz);
    a<<<ceil( dim*dim / 256),256>>>(c_A,c_B,c_C,dim);
    b<<<ceil(dim / 256),256>>>(c_A,c_B,c_C,dim);
    c<<<ceil(dim / 256),256>>>(c_A,c_B,c_C,dim);
    
    hipFree(c_A);
    hipFree(c_B);
    hipFree(c_C);
}

int main(){
    stub();
    return 0;
}