#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include "../../comm/hip/hip_runtime_api.h"

__global__ void a(float *a,float * b,float *c,size_t n) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if(i > n*n) return;
    a[i]=b[i]+c[i];
}


__global__ void b(float *a,float * b,float *c,size_t n) {
    int offset=blockIdx.x*blockDim.x+threadIdx.x;
    if(offset > n) return;
    for(int i=0;i<n;++i){
        a[i+offset]=b[i+offset]+c[i+offset];
    }
}


__global__ void c(float *a,float * b,float *c,size_t n) {
    int offset=blockIdx.x*blockDim.x+threadIdx.x;
    if(offset > n) return;
    for(int i=0;i<n;++i){
        a[i*n+offset]=b[i*n+offset]+c[i*n+offset];
    }
}
template<typename T>
void rand_array(T * array,size_t len){
    for(int i=0;i<len;++i){
        array[i]=((T)rand())/RAND_MAX;
    }
}
bool check(float* A,float* B,float * C,size_t dim){
    float* tmp=(float*)malloc(dim*dim*sizeof(float));
    for(int i=0;i<dim;++i){
        for(int j=0;j<dim;++j){
            tmp[i*dim+j]=B[i*dim+j]+C[i*dim+j];
            if(fabs(tmp[i*dim+j]-A[i*dim+j])>1e-10){ 
                free(tmp);
                return false;
            }
        }
    }
    free(tmp);
    return true;
}
int stub(){
    size_t dim=100;
    size_t siz=dim*dim*sizeof(float);
    float* B=(float*)malloc(siz);
    float* A=(float*)malloc(siz);
    float* C=(float*)malloc(siz);
    if(B==nullptr || A==nullptr || C==nullptr)
        return -1;
    float* c_A,*c_B,*c_C;
    rand_array(B,dim);
    rand_array(C,dim);    
    checkCudaErrors(hipMalloc((void**)&c_A, siz));
    checkCudaErrors(hipMalloc((void**)&c_B, siz));
    checkCudaErrors(hipMalloc((void**)&c_C, siz));
    checkCudaErrors(hipMemcpy(c_B, B, siz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(c_C, C, siz, hipMemcpyHostToDevice));
    a<<<ceil( dim*dim / 256),256>>>(c_A,c_B,c_C,dim);
    checkCudaErrors(hipMemcpy(A, c_A, siz, hipMemcpyDeviceToHost));
    if(!check(A,B,C,dim)){
        std::cout<<"A kernal err\n";
        return -1;
    } 
    b<<<ceil(dim / 256),256>>>(c_A,c_B,c_C,dim);
    checkCudaErrors(hipMemcpy(A, c_A, siz, hipMemcpyDeviceToHost));
    if(!check(A,B,C,dim)){
        std::cout<<"B kernal err\n";
        return -1;
    } 
    c<<<ceil(dim / 256),256>>>(c_A,c_B,c_C,dim);
    checkCudaErrors(hipMemcpy(A, c_A, siz, hipMemcpyDeviceToHost));
    if(!check(A,B,C,dim)){
        std::cout<<"C kernal err\n";
        return -1;
    } 
    checkCudaErrors(hipFree(c_A));
    checkCudaErrors(hipFree(c_B));
    checkCudaErrors(hipFree(c_C));
    return 0;
}

int main(){
    return stub();
}